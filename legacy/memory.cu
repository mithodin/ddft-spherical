#include <stdio.h>
#include "config.h"
#include "data.h"
#include "debug.h"
#include "dfts.h"

__host__ void dfts_init_kernelconfig(DFTS_kernelconfig *kc, size_t num_bins){
        kc->blocks_1 = (num_bins+THREADS-1)/THREADS;
        kc->threads = THREADS;
	kc->blocks_2.x = kc->blocks_1;
	kc->blocks_2.y = 2;
	kc->blocks_2.z = 1;
	kc->blocks_wd.x = kc->blocks_1;
	kc->blocks_wd.y = 2;
	kc->blocks_wd.z = dfts_num_wd;
}

extern "C" {
__host__ void dfts_sync(DFTS_t conf){
	do_check( hipMemcpy(conf->_self,conf,sizeof(DFTS_conf),hipMemcpyHostToDevice) == hipSuccess );
	hipDeviceSynchronize();
}

__host__ DFTS_t dfts_init(size_t num_bins, double dr, size_t bins_sphere){
	DFTS_t conf = (DFTS_t)malloc(sizeof(DFTS_conf));
	conf->num_bins = num_bins;
	conf->bins_sphere = bins_sphere;
	conf->dr = dr;
	conf->radius_sphere = dr*bins_sphere;
	conf->chemical_potential = 0.0;
	conf->selfinteraction = full;
	dfts_init_kernelconfig(&(conf->kc),num_bins);
	do_check( hipMalloc(&(conf->results),NUM_RESULTS*sizeof(double)) == hipSuccess );
	do_check( hipMalloc(conf->density_sum,2*num_bins*sizeof(double)) == hipSuccess );
	conf->density_sum[1] = conf->density_sum[0]+num_bins;
	do_check( hipMalloc(&(conf->min_mask),num_bins*sizeof(double)) == hipSuccess );
	do_check( hipMalloc(&(conf->potential),num_bins*sizeof(double)) == hipSuccess );
	do_check( hipMalloc(&(conf->grad_potential),num_bins*sizeof(double)) == hipSuccess );
	do_check( hipMalloc(conf->buffer,NUM_BUFFERS*num_bins*sizeof(double)) == hipSuccess );
	for(int i=1;i<NUM_BUFFERS;++i){
		conf->buffer[i] = conf->buffer[0]+i*num_bins;
	}
	do_check( hipMalloc(conf->weighted_density[0],2*dfts_num_wd*num_bins*sizeof(double)) == hipSuccess );
	conf->weighted_density[1][0] = conf->weighted_density[0][0]+dfts_num_wd*num_bins;
	for(int i=1;i<dfts_num_wd;++i){
		conf->weighted_density[0][i] = conf->weighted_density[0][0]+i*num_bins;
		conf->weighted_density[1][i] = conf->weighted_density[1][0]+i*num_bins;
	}

	do_check( hipMalloc(conf->psi[0],2*dfts_num_wd*num_bins*sizeof(double)) == hipSuccess );
	conf->psi[1][0] = conf->psi[0][0]+dfts_num_wd*num_bins;
	for(int i=1;i<dfts_num_wd;++i){
		conf->psi[0][i] = conf->psi[0][0]+i*num_bins;
		conf->psi[1][i] = conf->psi[1][0]+i*num_bins;
	}
	do_check( hipMalloc(conf->density,2*num_bins*sizeof(double)) == hipSuccess );
	conf->density[1] = conf->density[0]+num_bins;
	do_check( hipMalloc(conf->gradient,2*num_bins*sizeof(double)) == hipSuccess );
	conf->gradient[1] = conf->gradient[0]+num_bins;
	do_check( hipMalloc(&(conf->_self),sizeof(DFTS_conf)) == hipSuccess );
	do_check( hipMemcpy(conf->_self,conf,sizeof(DFTS_conf),hipMemcpyHostToDevice) == hipSuccess );
	return conf;
}

__host__ void dfts_destroy(DFTS_t conf){
	hipFree(conf->_self);
	hipFree(conf->gradient[0]);
	hipFree(conf->density[0]);
	hipFree(conf->weighted_density[0][0]);
	hipFree(conf->psi[0][0]);
	hipFree(conf->buffer[0]);
	hipFree(conf->grad_potential);
	hipFree(conf->min_mask);
	hipFree(conf->potential);
	hipFree(conf->density_sum[0]);
	hipFree(conf->results);
	free(conf);
}

}//end extern "C"
